#include "hip/hip_runtime.h"
// GPU kernels and wrappers for shared state methods.

#include <backends/event.hpp>
#include <backends/event_stream_state.hpp>

#include <arbor/gpu/gpu_api.hpp>
#include <arbor/gpu/gpu_common.hpp>

namespace arb {
namespace gpu {

namespace kernel {

// Vector/scalar addition: x[i] += v ∀i
template <typename T>
__global__ void add_scalar(unsigned n,
                           T* __restrict__ const x,
                           arb_value_type v) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        x[i] += v;
    }
}

__global__ void take_samples_impl(const raw_probe_info* __restrict__ const begin_marked,
                                  const raw_probe_info* __restrict__ const end_marked,
                                  const arb_value_type time,
                                  arb_value_type* __restrict__ const sample_time,
                                  arb_value_type* __restrict__ const sample_value) {
    const unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    const unsigned nsamples = end_marked - begin_marked;
    if (i<nsamples) {
        const auto p = begin_marked+i;
        sample_time[p->offset] = time;
        sample_value[p->offset] = p->handle? *p->handle: 0;
    }
}

} // namespace kernel

void add_scalar(std::size_t n, arb_value_type* data, arb_value_type v) {
    launch_1d(n, 128, kernel::add_scalar<arb_value_type>, n, data, v);
}

void take_samples_impl(const event_stream_state<raw_probe_info>& s,
                       const arb_value_type& time, arb_value_type* sample_time, arb_value_type* sample_value) {
    launch_1d(s.size(), 128, kernel::take_samples_impl, s.begin_marked, s.end_marked, time, sample_time, sample_value);
}

} // namespace gpu
} // namespace arb
